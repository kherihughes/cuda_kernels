#include <stdio.h>
#include <stdlib.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <cstdio>
#include "include/cuda_utils.h"
#include "include/convolution_ops.h"

#define H 1024     // Input height
#define W 1024     // Input width
#define C 3        // Number of input channels
#define FH 3       // Filter height
#define FW 3       // Filter width
#define K 64       // Number of filters (output channels)
#define P 1        // Padding

// Error checking macro
#define checkCUDNN(expression)                               \
{                                                           \
    hipdnnStatus_t status = (expression);                    \
    if (status != HIPDNN_STATUS_SUCCESS) {                   \
        fprintf(stderr, "Error on line %d: %s\n",           \
                __LINE__, hipdnnGetErrorString(status));     \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
}

int main() {
    // Define tensor dimensions
    int input_n = 1;    // Batch size
    int input_c = C;
    int input_h = H;
    int input_w = W;

    int output_n = 1;
    int output_c = K;
    int output_h = H;   // Assuming stride=1 and padding=P
    int output_w = W;

    int filter_k = K;
    int filter_c = C;
    int filter_h = FH;
    int filter_w = FW;

    // Allocate host memory
    size_t input_size = input_n * input_c * input_h * input_w * sizeof(float);
    size_t filter_size = filter_k * filter_c * filter_h * filter_w * sizeof(float);
    size_t output_size = output_n * output_c * output_h * output_w * sizeof(float);

    float *h_I = (float *)malloc(input_size);
    float *h_F = (float *)malloc(filter_size);
    float *h_O = (float *)malloc(output_size);

    // Initialize input tensor I
    for (int c = 0; c < C; c++)
        for (int y = 0; y < H; y++)
            for (int x = 0; x < W; x++)
                h_I[(c * H + y) * W + x] = c * (x + y);

    // Initialize filter tensor F with reversed indices (pre-flip)
    for (int k = 0; k < K; k++)
        for (int c = 0; c < C; c++)
            for (int j = 0; j < FH; j++)
                for (int i = 0; i < FW; i++) {
                    int fi = FW - 1 - i;
                    int fj = FH - 1 - j;
                    h_F[(((k * C + c) * FH + fj) * FW) + fi] = (c + k) * (i + j);
                }

    // Allocate device memory
    float *d_I, *d_F, *d_O;
    hipMalloc((void **)&d_I, input_size);
    hipMalloc((void **)&d_F, filter_size);
    hipMalloc((void **)&d_O, output_size);

    // Copy data to device memory
    hipMemcpy(d_I, h_I, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_F, h_F, filter_size, hipMemcpyHostToDevice);

    // Create cuDNN handle
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    // Create tensor descriptors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(
        input_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        input_n,
        input_c,
        input_h,
        input_w
    ));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(
        output_descriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        output_n,
        output_c,
        output_h,
        output_w
    ));

    // Create filter descriptor
    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(
        filter_descriptor,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        filter_k,
        filter_c,
        filter_h,
        filter_w
    ));

    // Create convolution descriptor
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(
        convolution_descriptor,
        P, P,   // padding
        1, 1,   // stride
        1, 1,   // dilation
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT
    ));
    
    
    hipdnnConvolutionFwdAlgo_t convolution_algorithm;

    // Choose the best convolution algorithm
    int returnedAlgoCount = 0;
    hipdnnConvolutionFwdAlgoPerf_t algoPerf;
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(
        cudnn,
        input_descriptor,
        filter_descriptor,
        convolution_descriptor,
        output_descriptor,
        1, // request one algorithm
        &returnedAlgoCount,
        &algoPerf
    ));
    convolution_algorithm = algoPerf.algo;


    // Get workspace size
    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn,
        input_descriptor,
        filter_descriptor,
        convolution_descriptor,
        output_descriptor,
        convolution_algorithm,
        &workspace_bytes
    ));

    // Allocate workspace memory
    void *d_workspace = NULL;
    hipMalloc(&d_workspace, workspace_bytes);

    // Set convolution scaling parameters
    float alpha = 1.0f;
    float beta = 0.0f;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Perform convolution
    hipEventRecord(start);
    checkCUDNN(hipdnnConvolutionForward(
        cudnn,
        &alpha,
        input_descriptor,
        d_I,
        filter_descriptor,
        d_F,
        convolution_descriptor,
        convolution_algorithm,
        d_workspace,
        workspace_bytes,
        &beta,
        output_descriptor,
        d_O
    ));
    hipEventRecord(stop);

    // Wait for convolution to finish
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(h_O, d_O, output_size, hipMemcpyDeviceToHost);

    // Compute checksum
    double checksum = 0.0;
    for (int idx = 0; idx < output_n * output_c * output_h * output_w; idx++) {
        checksum += (double)h_O[idx];
    }

    // Print checksum and kernel execution time
    printf("C3_checksum: %.6e\n", checksum);
    printf("C3_execution_time: %.3f ms\n", milliseconds);

    // Clean up
    hipFree(d_I);
    hipFree(d_F);
    hipFree(d_O);
    hipFree(d_workspace);

    free(h_I);
    free(h_F);
    free(h_O);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroy(cudnn);

    return 0;
}